#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define DataSize 1024
__global__ void Add(unsigned int *Da,int high,int width,int half)
{
	int tx = threadIdx.x;           
	int bx = blockIdx.x;			
	int bn = blockDim.x;
	//int gn = gridDim.x;
	int id = bx*bn+tx;				
	
	//for(int i=id;i<(high*width);i+=(bn*gn))
	  //Da[i] = 255 - Da[i];
	
	if (id < half) {
		Da[id] += Da[id + half];
	}
}
int main()
{
	FILE *fp = NULL;
	unsigned int high, width, offset;
	unsigned char *head;
	unsigned char *img; // �i��줸�A1 byte = 8 bits
	
	high = 0;
	width = 0;
	offset = 0;

	fp = fopen("lena.bmp","rb");

	fseek(fp, 10, SEEK_SET);
	fread(&offset, sizeof(unsigned int), 1, fp);

	fseek(fp, 18, SEEK_SET);
	fread(&width, sizeof(unsigned int), 1, fp);

	fseek(fp, 22, SEEK_SET);
	fread(&high, sizeof(unsigned int), 1, fp);

	img = (unsigned char*)malloc(sizeof(unsigned char)*(width*high));
	fseek(fp, offset, SEEK_SET);
	fread(img, sizeof(char), (width*high), fp);

	head =(unsigned char*)malloc(sizeof(unsigned char)*(offset));
	fseek(fp, 0, SEEK_SET);
	fread(head, sizeof(unsigned char), offset, fp);
	
	dim3 block(1024, 1, 1); // �@��block��1024��threads
	dim3 grid(256, 1, 1); // �@��grid��256��block
	
	unsigned int Dimg[512*512]; // CPU
	for (int j = 0; j < 512*512; j++) {
		Dimg[j] = img[j]; // �T�O�@��pixel�������A�Ӥp�N�e����0�A���|�Y��e�����ȡA�u�O�^��
	}
	
	unsigned int *Da; // GPU�A4ytes
	hipMalloc((void**)&Da, (sizeof(unsigned int)*(width*high))); // �t�mGPU�x�}�Ŷ�
	
	hipMemcpy(Da, Dimg, (sizeof(unsigned int)*(width*high)), hipMemcpyHostToDevice);
	
	int round = 0, half = 512*512;
	while (round < 18) { // lg(512*512)=lg(2^9*2^9)=lg(2^9)+lg(2^9)=9+9=18
		half /= 2;
		Add <<< grid, block >>> (Da,high,width,half); // �I�skernel
		hipDeviceSynchronize();
		round++;
	}
	
	hipMemcpy(Dimg, Da, (sizeof(unsigned int)*(width*high)), hipMemcpyDeviceToHost); // �ƻs��ƨ�GPU
	
	fclose(fp);
	
	Dimg[0] /= (512*512);
	printf("\n%3d\n", Dimg[0]);
}
