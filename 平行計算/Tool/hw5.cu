#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define DataSize 1024

// �C��ۭ�:(2^8)*(2^8)=2^16,�@�C���@��:��(2^9)��,�s�x�}�C��:(2^16)*(2^9)=2^25,�@��(2^25)*(2^9)*(2^9)=2^43 < 2^32 

__global__ void Add(unsigned long *Da,unsigned long *Db,int high,int width)
{
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int bn = blockDim.x;
	//int gn = gridDim.x;
	int id = bx*bn+tx;				
	
	//for(int i=id;i<(high*width);i+=(bn*gn))
	  //Da[i] = 255 - Da[i];
	
	unsigned long tmp = 0;
	
	for (int i = 0; i < 512; i++) { // 512*512���x�} 
		tmp += Da[bx*512+i] * Da[i*512+tx];
	}
	Db[bx*512+tx] = tmp;
	__syncthreads();
	
	int i = bn/2; // �C��block����threads�Ӱ�
	
	while (i != 0) {
		if (tx < i) {
		  Db[id] += Db[id + i];
		}
		__syncthreads();
		i /= 2;
	}
}
int main()
{
	FILE *fp = NULL;
	unsigned int high, width, offset;
	unsigned char *head;
	unsigned char *img; // �i��줸�A1 byte = 8 bits
	
	high = 0;
	width = 0;
	offset = 0;

	fp = fopen("lena.bmp","rb");

	fseek(fp, 10, SEEK_SET);
	fread(&offset, sizeof(unsigned int), 1, fp);

	fseek(fp, 18, SEEK_SET);
	fread(&width, sizeof(unsigned int), 1, fp);

	fseek(fp, 22, SEEK_SET);
	fread(&high, sizeof(unsigned int), 1, fp);
	
	img = (unsigned char*)malloc(sizeof(unsigned char)*(width*high));
	fseek(fp, offset, SEEK_SET);
	fread(img, sizeof(char), (width*high), fp);

	head =(unsigned char*)malloc(sizeof(unsigned char)*(offset));
	fseek(fp, 0, SEEK_SET);
	fread(head, sizeof(unsigned char), offset, fp);
	
	unsigned int nthread, nblock;
	
	if(width > 1024) { // �ۭ��n����width*high�A��thread���פ���W�L1024
        nthread = 1024; 
        nblock = width * high / 1024;
    } else {
        nthread = width;
        nblock = high;
    }
	dim3 block(nthread, 1, 1); // �@��block��1024��threads
	dim3 grid(nblock, 1, 1); // �@��grid��256��block
	
	unsigned long Dimg[512*512]; // CPU
	for (int j = 0; j < 512*512; j++) { // �t�mCPU�x�}�Ŷ�
		Dimg[j] = img[j]; // �T�O�@��pixel�������A�Ӥp�N�e����0�A���|�Y��e�����ȡA�u�O�^��
	}
	
	unsigned long *Da; // GPU�A4ytes
	hipMalloc((void**)&Da, (sizeof(unsigned long)*(width*high))); // �t�mGPU�x�}�Ŷ�
	
	hipMemcpy(Da, Dimg, (sizeof(unsigned long)*(width*high)), hipMemcpyHostToDevice); //�ƻs��ƨ�GPU
	
	unsigned long *Db; // GPU�A4ytes
	hipMalloc((void**)&Db, (sizeof(unsigned long)*(width*high))); // �t�mGPU�x�}�Ŷ�
	
	Add <<< grid, block >>> (Da,Db,high,width); // �I�skernel
	hipDeviceSynchronize();
	
	hipMemcpy(Dimg, Db, (sizeof(unsigned long)*(width*high)), hipMemcpyDeviceToHost); // �ƻs���(���G)�^CPU
	
	fclose(fp);
	
	unsigned long sum = 0;
	/*for (int i = 0; i < 512*512; i++) {
		sum += Dimg[i];
	}*/
	for (int i = 0; i < nthread*nblock; i += nthread) {
		sum += Dimg[i];
	}
	sum /= (512*512);
	
	printf("\n%3lu\n", sum);
}
