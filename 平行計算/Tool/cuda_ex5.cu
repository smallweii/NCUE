#include "hip/hip_runtime.h"
//一維陣列相加的範例程式

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DataSize 16

void GenerateNumbers(int *number, int size, int k)//隨機產生資料
{
     int i;
	   srand(k * time(NULL));
     for (i = 0; i < size; i++)
         number[i] = rand() % 100;
}

__global__ void Add_A(int *Da)//kernel function
{
	int tx = threadIdx.x;     //thread的x軸id
	int bx = blockIdx.x;	    //block的x軸id
	int bn = blockDim.x;		//block的x軸有幾個thread
	int id = bx*bn+tx;		//計算矩陣座標
	
	/*int i = DataSize/2;
	while (i != 0) {
	    if (id < i)
		    Da[id] += Da[id + i];
		__syncthreads();
		i /= 2;
	}*/
	
	int i = DataSize/2;
	for(int j = 0; j < 4; j++) { // log16=4
        if (id < i)
            Da[id] += Da[id + i];
        __syncthreads( );
		i /= 2;
    }
}

int main()
{
	int *Ha; //CPU
	int size = DataSize * sizeof(int);
	
	Ha = (int*)malloc(size);				//配置矩陣空間
	GenerateNumbers(Ha, DataSize, 2);		//產生矩陣資料

	/* dim3 由CUDA提供的三維向量型態 (X,Y,Z)
		CUDA限制每個block的thread上限為1024, (X*Y*Z)<=1024
		grid的block上限為65535, (X*Y)<=65535.  block最多2維而已
	*/	
	dim3 block(DataSize/2, 1, 1);			//配置thread維度、大小  
	dim3 grid(2, 1, 1);		//配置block維度、大小	
	
    int i;
	printf("A\n");
	for (i = 0; i < DataSize; i++)
		printf("%3d ", Ha[i]);
		
	int *Da; //GPU
	hipMalloc((void**)&Da, size);			//配置GPU矩陣空間
	hipMemcpy(Da, Ha, size, hipMemcpyHostToDevice);		//複製資料到GPU

	Add_A <<< grid, block >>> (Da);			//呼叫kernel
	hipDeviceSynchronize();
	
	hipMemcpy(Ha, Da, size, hipMemcpyDeviceToHost);		//複製資料(比較後的結果)回CPU
	
	printf("\n%3d\n", Ha[0]);

	//釋放記憶體空間
		free(Ha);
	
		hipFree(Da);
}
