#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define DataSize 1024
__global__ void Add(unsigned int *Da,int high,int width)
{
	int tx = threadIdx.x;           
	int bx = blockIdx.x;			
	int bn = blockDim.x;
	//int gn = gridDim.x;
	int id = bx*bn+tx;				
	
	//for(int i=id;i<(high*width);i+=(bn*gn))
	  //Da[i] = 255 - Da[i];
	
	int i = 1024/2; // �C��block����threads�Ӱ�
	
	while (i != 0) {
		if (tx < i) {
		  Da[id] += Da[id + i];
		}
		__syncthreads();
		i /= 2;
	}
}
int main()
{
	FILE *fp = NULL;
	unsigned int high, width, offset;
	unsigned char *head;
	unsigned char *img; // �i��줸�A1 byte = 8 bits
	
	high = 0;
	width = 0;
	offset = 0;

	fp = fopen("lena.bmp","rb");

	fseek(fp, 10, SEEK_SET);
	fread(&offset, sizeof(unsigned int), 1, fp);

	fseek(fp, 18, SEEK_SET);
	fread(&width, sizeof(unsigned int), 1, fp);

	fseek(fp, 22, SEEK_SET);
	fread(&high, sizeof(unsigned int), 1, fp);

	img = (unsigned char*)malloc(sizeof(unsigned char)*(width*high));
	fseek(fp, offset, SEEK_SET);
	fread(img, sizeof(char), (width*high), fp);

	head =(unsigned char*)malloc(sizeof(unsigned char)*(offset));
	fseek(fp, 0, SEEK_SET);
	fread(head, sizeof(unsigned char), offset, fp);
	
	dim3 block(1024, 1, 1); // �@��block��1024��threads
	dim3 grid(256, 1, 1); // �@��grid��256��block
	
	unsigned int Dimg[512*512]; // CPU
	for (int j = 0; j < 512*512; j++) {
		Dimg[j] = img[j]; // �T�O�@��pixel�������A�Ӥp�N�e����0�A���|�Y��e�����ȡA�u�O�^��
	}
	
	unsigned int *Da; // GPU�A4ytes
	hipMalloc((void**)&Da, (sizeof(unsigned int)*(width*high))); // �t�mGPU�x�}�Ŷ�
	
	hipMemcpy(Da, Dimg, (sizeof(unsigned int)*(width*high)), hipMemcpyHostToDevice);
	
	Add <<< grid, block >>> (Da,high,width); // �I�skernel
	hipDeviceSynchronize();
	
	hipMemcpy(Dimg, Da, (sizeof(unsigned int)*(width*high)), hipMemcpyDeviceToHost); // �ƻs��ƨ�GPU
	
	fclose(fp);
	
	unsigned int sum = 0;
	for (int i = 0; i < 512*512; i += 1024) { // 256��block
		sum += Dimg[i];
	}
	sum /= (512*512);
	
	printf("\n%3d\n", sum);
}
