#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define DataSize 1024
__global__ void Add(unsigned char *Da,int high,int width)
{
	int tx = threadIdx.x;           
	int bx = blockIdx.x;			
	int bn = blockDim.x;
	int gn = gridDim.x;
	int id = bx*bn+tx;				

	for(int i=id;i<(high*width);i+=(bn*gn))
		Da[i] = 255 - Da[i];

}
int main()
{
	FILE *fp = NULL;
	unsigned int high, width, offset;
	unsigned char *head;
	unsigned char *img;
	
	high = 0;
	width = 0;
	offset = 0;

	fp = fopen("lena.bmp","rb");

	fseek(fp, 10, SEEK_SET);
	fread(&offset, sizeof(unsigned int), 1, fp);

	fseek(fp, 18, SEEK_SET);
	fread(&width, sizeof(unsigned int), 1, fp);

	fseek(fp, 22, SEEK_SET);
	fread(&high, sizeof(unsigned int), 1, fp);

	img = (unsigned char*)malloc(sizeof(unsigned char)*(width*high));
	fseek(fp, offset, SEEK_SET);
	fread(img, sizeof(char), (width*high), fp);

	head =(unsigned char*)malloc(sizeof(unsigned int)*(offset));
	fseek(fp, 0, SEEK_SET);
	fread(head, sizeof(unsigned char), offset, fp);
	
	dim3 block(1024, 1, 1);
	dim3 grid(2, 1, 1);
	
	unsigned char *Da;
	hipMalloc((void**)&Da, (sizeof(unsigned char)*(width*high)));
	
	hipMemcpy(Da, img, (sizeof(unsigned char)*(width*high)), hipMemcpyHostToDevice);
	
	Add <<< grid, block >>> (Da,high,width);
	hipDeviceSynchronize();
	
	hipMemcpy(img, Da, (sizeof(unsigned char)*(width*high)), hipMemcpyDeviceToHost);
	
	fclose(fp);

	fp = fopen("lena2.bmp","wb+");
	fwrite(head, sizeof(unsigned char), offset, fp);
	fwrite(img, sizeof(unsigned char), (width*high), fp);
	fclose(fp);

}


