#include "hip/hip_runtime.h"
//一維陣列相加的範例程式

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DataSize 16*16 // 16*16矩陣

void GenerateNumbers(int *number, int size, int k)//隨機產生資料
{
     int i;
	   srand(k * time(NULL));
     for (i = 0; i < size; i++)
         number[i] = rand() % 100;
}

__global__ void kernel(int *Da, int *Db, int *Dc)//kernel function
{
	int tx = threadIdx.x;      //thread的x軸id
	int ty = threadIdx.y;      //thread的y軸id
	int tmp = 0;
	int n = 16;

	for (int i = 0; i < n; i++)
        tmp += Da[ty*n+i] * Db[i*n+tx];
    Dc[ty*n+tx] = tmp;
}

int main()
{
	int *Ha, *Hb, *Hc; //CPU
	int size = DataSize * sizeof(int);
	
	Ha = (int*)malloc(size);				//配置矩陣空間
	Hb = (int*)malloc(size);				//配置矩陣空間
	Hc = (int*)malloc(size);				//配置矩陣空間
	
	GenerateNumbers(Ha, DataSize, 2);		//產生矩陣資料
	GenerateNumbers(Hb, DataSize, 6);		//產生矩陣資料

	
	/* dim3 由CUDA提供的三維向量型態 (X,Y,Z)
		CUDA限制每個block的thread上限為1024, (X*Y*Z)<=1024
		grid的block上限為65535, (X*Y)<=65535.  block最多2維而已
	*/	
	
	dim3 block(16, 16, 1);	 //配置thread維度、大小  
	dim3 grid(1, 1, 1);	     //配置block維度、大小
	
	int *Da, *Db, *Dc; //GPU
	hipMalloc((void**)&Da, size);			//配置GPU矩陣空間
	hipMalloc((void**)&Db, size);			//配置GPU矩陣空間
	hipMalloc((void**)&Dc, size);			//配置GPU矩陣空間

	hipMemcpy(Da, Ha, size, hipMemcpyHostToDevice);		//複製資料到GPU
	hipMemcpy(Db, Hb, size, hipMemcpyHostToDevice);		//複製資料到GPU

	kernel <<< grid, block >>> (Da, Db, Dc);			//呼叫kernel
	hipDeviceSynchronize();

	hipMemcpy(Hc, Dc, size, hipMemcpyDeviceToHost);		//複製資料(比較後的結果)回CPU

	int i;
	printf("A\n");
	for (i = 0; i < DataSize; i++) {
		printf("%3d ", Ha[i]);
		if (i % 16 == 15)
			printf("\n");
	}
	
	printf("\nB\n");
	for (i = 0; i < DataSize; i++) {
		printf("%3d ", Hb[i]);
		if (i % 16 == 15)
			printf("\n");
	}
		
	printf("\nC\n");
	for (i = 0; i < DataSize; i++) {
		printf("%3d ", Hc[i]);
		if (i % 16 == 15)
			printf("\n");
	}
	printf("\n");
	
	//釋放記憶體空間
		free(Ha); free(Hb); free(Hc);
	
		hipFree(Da); hipFree(Db); hipFree(Dc);
}
