#include "hip/hip_runtime.h"
//一維陣列相加的範例程式

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DataSize 16

void GenerateNumbers(int *number, int size, int k)//隨機產生資料
{
     int i;
	   srand(k * time(NULL));
     for (i = 0; i < size; i++)
         number[i] = rand() % 100;
}

__global__ void Compare_A(int *Da, int *Dc)//kernel function
{
	int tx = threadIdx.x;     //thread的x軸id
	int bx = blockIdx.x;	    //block的x軸id
	int bn = blockDim.x;		//block的x軸有幾個thread
	int id = bx*bn+tx;		//計算矩陣座標
	
	//Dc[id] = Da[id] + Db[id];
	if (Da[id*2] >= Da[id*2+1])
		Dc[id] = Da[id*2];
	else
		Dc[id] = Da[id*2+1];
}

int main()
{
	int *Ha, *Hc; //CPU
	int size = DataSize * sizeof(int);
	
	Ha = (int*)malloc(size);				//配置矩陣空間
	//Hb = (int*)malloc(size);				//配置矩陣空間
	Hc = (int*)malloc(size);				//配置矩陣空間
	
	GenerateNumbers(Ha, DataSize, 2);		//產生矩陣資料
	//GenerateNumbers(Hb, DataSize, 6);		//產生矩陣資料

	
	/* dim3 由CUDA提供的三維向量型態 (X,Y,Z)
		CUDA限制每個block的thread上限為1024, (X*Y*Z)<=1024
		grid的block上限為65535, (X*Y)<=65535.  block最多2維而已
	*/	
	dim3 block(DataSize/2, 1, 1);			//配置thread維度、大小  
	dim3 grid(2, 1, 1);						//配置block維度、大小	

	
	
	int *Da, *Dc; //GPU
	hipMalloc((void**)&Da, size);			//配置GPU矩陣空間
	//hipMalloc((void**)&Db, size);			//配置GPU矩陣空間
	hipMalloc((void**)&Dc, size);			//配置GPU矩陣空間

	hipMemcpy(Da, Ha, size, hipMemcpyHostToDevice);		//複製資料到GPU
	//hipMemcpy(Db, Hb, size, hipMemcpyHostToDevice);		//複製資料到GPU

	Compare_A <<< grid, block >>> (Da, Dc);			//呼叫kernel
	hipDeviceSynchronize();

	hipMemcpy(Hc, Dc, size, hipMemcpyDeviceToHost);		//複製資料(比較後的結果)回CPU

	int i;
	printf("A\n");
	for (i = 0; i < DataSize; i++)
		printf("%3d ", Ha[i]);
	
	//printf("\nB\n");
	//for (i = 0; i < DataSize; i++)
		//printf("%3d ", Hb[i]);
	printf("\nC\n");
	for (i = 0; i < DataSize/2; i++)
		printf("%3d ", Hc[i]);
	printf("\n");

	
	//釋放記憶體空間
		free(Ha); free(Hc);
	
		hipFree(Da); hipFree(Dc);
}
