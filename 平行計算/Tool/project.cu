#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define DataSize 1024
__global__ void Mirror(unsigned int *Da, unsigned int *Dc, int high,int width)
{
	int tx = threadIdx.x;           
	int bx = blockIdx.x;			
	int bn = blockDim.x;
	int id = bx*bn+tx;
        
    Dc[id] = Da[bx * bn + bn - 1 - tx];
}
__global__ void Filter_func(unsigned int *Da, unsigned int *Dc, int high,int width, unsigned int *Filter, int filtersum)
{
	int tx = threadIdx.x;           
	int bx = blockIdx.x;			
	int bn = blockDim.x;
	int id = bx*bn+tx;
 	
 	__shared__ unsigned int share[512 * 3];
 	
    if(bx == 0 || bx == bn - 1){
        if(bx == 0){
            share[tx + bn] = Da[id];
            share[tx + 2* bn] = Da[id + bn];
        }else{
            share[tx + bn] = Da[id];
            share[tx] = Da[id - bn]; 
        }
    }else{
        share[tx] = Da[id - bn];
        share[tx + bn] = Da[id];
        share[tx + 2 * bn] = Da[id + bn];
    }
    __syncthreads();
        
    int sum = 0;
        
    if(bx != 0 && tx % 512 != 0) // ���W�S�W�X�d��
	    sum += Filter[0] * share[tx - 1];	

    if(bx != 0) // �W���S�W�X�d��
        sum += Filter[1] * share[tx];

    if(bx != 0 && (tx + 1) % 512 != 0) // �k�W�S�W�X�d��
        sum += Filter[2] * share[tx + 1];

    if(tx % 512 != 0) // ����S�W�X�d��
        sum += Filter[3] * share[tx + bn - 1];

    sum += Filter[4] * share[tx + bn];

    if((tx + 1) % 512 != 0) // �k��S�W�X�d��
        sum += Filter[5] * share[tx + bn + 1];

    if(bx != bn - 1 && tx % 512 != 0) // ���U�S�W�X�d�� 
        sum += Filter[6] * share[tx + 2 * bn - 1];

    if(bx != bn - 1) // �U���S�W�X�d��
        sum += Filter[7] * share[tx + 2 * bn];

    if(bx != bn - 1 && (tx + 1) % 512 != 0) // �k�U�S�W�X�d��
        sum += Filter[8] * share[tx + 2 * bn + 1];
     
    Dc[id] = sum / filtersum;
}        
int main()
{
	FILE *fp = NULL;
	unsigned int high, width, offset, nthread, nblock;
	unsigned char *head;
	unsigned char *img;
    unsigned int *da;
    unsigned int filter[9] = {1,2,1,2,4,2,1,2,1};
    char c, sel;
        
    printf("What do you want to do?\n(a)Filter the picture\n(b)Mirror the picture\n");
    scanf("%c", &sel);
    getchar();
    
    if(sel == 'A' || sel == 'a'){
        printf("Do you want to input filter values by yourself? (Y/N)\n");
        scanf("%c", &c);
    	getchar();
    
        if(c == 'Y' || c == 'y'){
            printf("Please input 9 filter values:\n");
            for(int i = 0; i < 9; ++i)
                scanf("%d", &filter[i]);
                getchar();
        }
    }
	high = 0;
	width = 0;
	offset = 0;

	fp = fopen("lena.bmp","rb"); // rb�Oread binary

	fseek(fp, 10, SEEK_SET); // ���в���fp�q�Y�⪺��10��byte�A�]��offset��bheader����10~13��byte
	fread(&offset, sizeof(unsigned int), 1, fp); // �qfpŪ1��unsigned int���j�p�s��offset

	fseek(fp, 18, SEEK_SET); // ���в���fp�q�Y�⪺��18��byte�A�]��width��bheader����18~21��byte
	fread(&width, sizeof(unsigned int), 1, fp); // �qfpŪ1��unsigned int���j�p�s��width

	fseek(fp, 22, SEEK_SET); // ���в���fp�q�Y�⪺��22��byte�A�]��high��bheader����22~25��byte
	fread(&high, sizeof(unsigned int), 1, fp); // �qfpŪ1��unsigned int���j�p�s��high

	img = (unsigned char*)malloc(sizeof(unsigned char)*(width*high)); // �t�m�@��width*high��byte�Ƥj�p��img(unsigned char�O1byte)
    da = (unsigned int*)malloc(sizeof(unsigned int)*(width*high));
	fseek(fp, offset, SEEK_SET); // ���в���fp�q�Y��offset��byte����m�A�]��offset�s����m�O�}�l�spixel����m
	fread(img, sizeof(char), (width*high), fp); // �qfpŪwidth*high��char���j�p(=unsigned char���j�p)�s��img

	head =(unsigned char*)malloc(sizeof(unsigned char)*(offset)); // �t�m�@��offset��byte�Ƥj�p��img
	fseek(fp, 0, SEEK_SET); // ���в���fp���@�}�l�A�q�Y����m��offset����m�O����Y��
	fread(head, sizeof(unsigned char), offset, fp); // �qfpŪoffset��unsigned char���j�p(=unsigned char���j�p)�s��head
	
    if(width > 1024){ // ���_�ӭn����width*high�Athread���פ���W�L1024
        nthread = 1024; 
        nblock = high * width / 1024;
    }else{
        nthread = width;
        nblock = high;
    }
	dim3 block(nthread, 1, 1); 
    dim3 grid(nblock, 1, 1);
	
    for(int i = 0; i < high * width; ++i)
        da[i] = img[i];

    int filtersum = 0;
    for(int i = 0; i < 9; ++i)
        filtersum += filter[i];

	unsigned int *Da;
	hipMalloc((void**)&Da, (sizeof(unsigned int)*(width*high)));
    unsigned int *Dc;
	hipMalloc((void**)&Dc, (sizeof(unsigned int)*(width*high)));
    unsigned int *Filter;
	hipMalloc((void**)&Filter, (sizeof(unsigned int)*9));
	
	hipMemcpy(Da, da, (sizeof(unsigned int)*(width*high)), hipMemcpyHostToDevice);
    if(sel == 'A' || sel == 'a'){
	    Filter_func <<< nblock, nthread >>> (Da,Dc,high,width,Filter,filtersum);
        hipMemcpy(Filter, filter, (sizeof(unsigned int)*9), hipMemcpyHostToDevice);
    }else if(sel == 'B' || sel == 'b'){
        Mirror <<< nblock, nthread >>> (Da,Dc,high,width);
    }
	hipDeviceSynchronize();
    hipMemcpy(da, Dc, (sizeof(unsigned int)*(width*high)), hipMemcpyDeviceToHost);	

    for(int i = 0; i < high * width; ++i)
        img[i] = da[i];
      
	fp = fopen("lena3.bmp","wb+");
	fwrite(head, sizeof(unsigned char), offset, fp);
	fwrite(img, sizeof(unsigned int), (width*high), fp);
	fclose(fp);
}
